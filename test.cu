
#include <hip/hip_runtime.h>
struct payoffStruct;
struct yieldTermStruct;
struct blackVolStruct;
struct blackScholesMertStruct;
struct optionStruct;
struct blackCalcStruct;
struct normalDistStruct;
struct optionInputStruct;
typedef unsigned long size_t;
struct payoffStruct {
int type;
float strike;};
typedef struct payoffStruct payoffStruct;
struct yieldTermStruct {
float timeYearFraction;
float forward;
float compounding;
float frequency;
float intRate;};
typedef struct yieldTermStruct yieldTermStruct;
struct blackVolStruct {
float timeYearFraction;
float following;
float volatility;};
typedef struct blackVolStruct blackVolStruct;
struct blackScholesMertStruct {
float x0;
yieldTermStruct dividendTS;
yieldTermStruct riskFreeTS;
blackVolStruct blackVolTS;};
typedef struct blackScholesMertStruct blackScholesMertStruct;
struct optionStruct {
payoffStruct payoff;
float yearFractionTime;
blackScholesMertStruct pricingEngine;};
typedef struct optionStruct optionStruct;
struct blackCalcStruct {
float strike;
float forward;
float stdDev;
float discount;
float variance;
float d1;
float d2;
float alpha;
float beta;
float DalphaDd1;
float DbetaDd2;
float n_d1;
float cum_d1;
float n_d2;
float cum_d2;
float x;
float DxDs;
float DxDstrike;};
typedef struct blackCalcStruct blackCalcStruct;
struct normalDistStruct {
float average;
float sigma;
float denominator;
float derNormalizationFactor;
float normalizationFactor;};
typedef struct normalDistStruct normalDistStruct;
struct optionInputStruct {
int type;
float strike;
float spot;
float q;
float r;
float t;
float vol;
float value;
float tol;};
typedef struct optionInputStruct optionInputStruct;